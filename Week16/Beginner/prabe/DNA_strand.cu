
#include <hip/hip_runtime.h>
#include <stdio.h>

#define DNA_LENGTH 2062336
#define THREADS	1024 	//1024 max threads per block 
#define BLOCKS	1024

__global__ void DNA_func(int *org, int *newar, int n)
{
	//each thread in each block determines which part of the array it is working on.
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	
	//catch if there are more threads than there is data to be worked on.(to prevent memory access issues)
	if (i < n)
	{
		switch(org[i]) {
			case 'A':
			case 'a':
			newar[i] = 'T';
			break;
			
			case 'T':
			case 't':
			newar[i] = 'A';
			break;
			
			case 'G':
			case 'g':
			newar[i] = 'C';
			break;
			
			case 'C':
			case 'c':
			newar[i] = 'G';
			break;
			
			default:
			newar[i] = 'N';
			break;
		}
	}
}

int main()
{
	hipFree(0);//init cuda(its lazy and does not startup until needed)
	
	int *Original, *Newar;//declaration of array for host(cpu)
	int *d_Original, *d_Newar; //declaration of arrays for device(gpu)
	int c,n;//char from file, number of chars pulled
	c = 0;
	n = 0;
	
	Original = (int *)malloc(DNA_LENGTH*sizeof(int));//alcoate memory of arrays in host(cpu)
	Newar = (int *)malloc(DNA_LENGTH*sizeof(int));
	
	hipMalloc( &d_Original, DNA_LENGTH*sizeof(int));//init of arrays in device(gpu)
	hipMalloc( &d_Newar, DNA_LENGTH*sizeof(int));
	
	FILE *fp = fopen("dna.txt", "r");
	if (fp == NULL){
	 printf("File DNE");
	 return 1;
	 }
	 c = fgetc(fp);
	while((c != EOF)&&(n < DNA_LENGTH)){
			if ((c == 'A')||(c == 'a')||(c == 'T')||(c == 't')||(c == 'G')||(c == 'g')||(c == 'C')||(c == 'c')){
			Original[n] = c;
			Newar[n] = c;
		
			n++;
			}
		c = fgetc(fp);
	 	}
	 	fclose(fp);
	hipMemcpy( d_Original, Original, DNA_LENGTH*sizeof(int), hipMemcpyHostToDevice );//copy array data to device(gpu)
	hipMemcpy( d_Newar, Newar, DNA_LENGTH*sizeof(int), hipMemcpyHostToDevice );
	
	DNA_func<<< THREADS, BLOCKS >>>(d_Original, d_Newar, n-1);//init gpu kernel (the magic happens here!)
	
	hipMemcpy( Newar, d_Newar, DNA_LENGTH*sizeof(int), hipMemcpyDeviceToHost );//copy array output data back to host (cpu)
	FILE *fp2 = fopen("dna_out.txt", "w");

if (n <= 25){
	for(int i = 0; i<(n-1); i++)
		{
			printf("%c %c \n",Original[i],Newar[i]);//prints out the output if it is small...
		}
}
	for(int i = 0; i<(n-1); i++)
		{
			fprintf(fp2,"%c %c \n",Original[i],Newar[i]);//outputs to a file so you are not waiting for the terminal forever.
		}
		fclose(fp2);
	free(Original);
	free(Newar);
	hipFree(d_Original);
	hipFree(d_Newar);
	return 0;
}
