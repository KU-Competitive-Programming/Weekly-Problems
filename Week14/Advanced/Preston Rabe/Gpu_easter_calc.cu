
#include <hip/hip_runtime.h>
#include <stdio.h>
/*
	A GPU implemenation of finding the dates of easter using a Nvidia Cuda GPU with Cuda C.
	
	nvcc -o easter Gpu_easter_calc.cu
	
	Preston Rabe
	
*/

#define YEARS	16384	//years = treads * blocks (16384 years after year 2000!)
#define THREADS	128 	//1024 max threads per block 
#define BLOCKS	128		


__global__ void easter(int *yr, int *mon, int *day, int n)//GPU task arguement 'n' is to catch if more threads are started than there is data to be worked on
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;//each thread in each block determines which part of the array it is working on.
	
	if (i < n)//catch if there are more threads than there is data to be worked on.(to prevent memory access issues)
	{
		//Computeing easter day/month (can't have many variables per thread, memory is limited.)
		int a = (yr[i]+2000) % 19;
		int b = (yr[i]+2000)>>2;
		int c = b/25+1;
		int d = (c*3)>>2;
		int e = ((a*19) - ((c*8+5)/25) +d +15)%30;
		e += (29578 -a -e *32) >> 10;
		e -= (((yr[i]+2000)%7)+b-d+e+2)%7;
		d = e>>5;
		//saveing day/month in to GPU memory arrays
		day[i] = e-d*31;
		mon[i] = d+3;
	}
}

int main()
{

	int *a, *b, *c;//declaration of array for host(cpu)
	int *d_a, *d_b, *d_c;//declaration of arrays for device(gpu)
	
	a = (int *)malloc(YEARS*sizeof(int));//alcoate memory of arrays in host(cpu)
	b = (int *)malloc(YEARS*sizeof(int));
	c = (int *)malloc(YEARS*sizeof(int));
	
	hipMalloc( &d_a, YEARS*sizeof(int));//init of arrays in device(gpu)
	hipMalloc( &d_b, YEARS*sizeof(int));
	hipMalloc( &d_c, YEARS*sizeof(int));
	
	for( int i = 0; i < YEARS; ++i )//init arrays values
	{
		a[i] = i;//years (after 2000)
		b[i] = 0;//month
		c[i] = 0;//days
	}
	
	hipMemcpy( d_a, a, YEARS*sizeof(int), hipMemcpyHostToDevice );//copy array data to device(gpu)
	hipMemcpy( d_b, b, YEARS*sizeof(int), hipMemcpyHostToDevice );
	hipMemcpy( d_c, c, YEARS*sizeof(int), hipMemcpyHostToDevice );
	
	easter<<< THREADS, BLOCKS >>>(d_a, d_b, d_c, YEARS);//init gpu kernel (the magic happens here!)
	
	hipMemcpy( b, d_b, YEARS*sizeof(int), hipMemcpyDeviceToHost );//copy array output data back to host (cpu)
	hipMemcpy( c, d_c, YEARS*sizeof(int), hipMemcpyDeviceToHost );
	
	for( int i = 0; i < YEARS; ++i)
		printf("%2d, %d,%d\n", c[i],b[i],i+2000);//print out data to terminal (day,month,year)
	
	free(a);
	free(b);
	free(c);//free memory in host(cpu)
	
	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);//free memory in device(gpu)
	
	return 0;//exit program
}
